#include "hip/hip_runtime.h"
#include "omp.h"
#include "stdio.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void
calcularPi( float *sum, int operaciones)
{
  int i = ((blockDim.x * blockIdx.x + threadIdx.x));
  sum[i] = 0;
	for(int j = 0; j < operaciones; j++){
    float aumento = (float)(2/((4*(i + j) + 1)*(4*(i + j) + 3)));
    sum[i] += aumento ;
	}
    
}
int main(void)
{
  // declarar  la cantidad de hilos segun la gpu
  hipError_t err = hipSuccess;
  int dev = 0;
  size_t size = sizeof(float);
	hipSetDevice(dev);
  hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	int threadsPerBlock = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
	threadsPerBlock = threadsPerBlock*2;
  int blocksPerGrid =   deviceProp.multiProcessorCount;
  float numIt = 4e9;
  int hilosTotales = blocksPerGrid*threadsPerBlock;
  int operacionPorHilo;
  size_t size_pi = sizeof(float) * hilosTotales;
  operacionPorHilo = (numIt > hilosTotales ) ? (int)(ceil(numIt/hilosTotales) ) : 1;
  float h_pi = 0.0;
  float *h_sum = (float*)malloc(size_pi);
  float *d_sum = NULL;
  err = hipMalloc((void **)&d_sum, size_pi);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device d_sum (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }
  err = hipMemcpy(d_sum, h_sum, size_pi, hipMemcpyHostToDevice);

  

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector pi from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  printf("Operaciones por Hilo %d\n",operacionPorHilo);
  calcularPi<<<blocksPerGrid, threadsPerBlock>>>(d_sum, operacionPorHilo);
  err = hipGetLastError();

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to launch calcularPi kernel (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_sum, d_sum, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        
        fprintf(stderr, "Failed to copy h_pi from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    for(int i = 0 ; i < hilosTotales; i ++){
        h_pi += h_sum[i];
    }

    printf("valor de pi %f\n", (h_pi)*4);
    return 0;

}