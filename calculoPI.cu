#include "hip/hip_runtime.h"
#include "omp.h"
#include "stdio.h"
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void
calcularPi(float *pi, int operaciones)
{
  int i = ((blockDim.x * blockIdx.x + threadIdx.x)*operaciones);
  if(threadIdx.x < 10 && blockIdx.x == 0){
    printf("valor de i%d\tblockDim%d\tblockId%d\threadhId%d\n",i, blockDim.x, blockIdx.x,threadIdx.x);
  }
	for(int j = 0; j < operaciones; j++){
    i = i + j;
    *pi = *pi + (2/((4*(i) + 1)*(4*(i) + 3)));
	}
    
}
int main(void)
{
  // declarar  la cantidad de hilos segun la gpu
  hipError_t err = hipSuccess;
  int dev = 0;
  size_t size = sizeof(float);
	hipSetDevice(dev);
  hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	int threadsPerBlock = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
	threadsPerBlock = threadsPerBlock*2;
  int blocksPerGrid =   deviceProp.multiProcessorCount;
  int numIt = 4e8;
  int hilosTotales = blocksPerGrid*threadsPerBlock;
  int operacionPorHilo;
  operacionPorHilo = (numIt > hilosTotales ) ? (int)(ceil(numIt/(float)hilosTotales) ) : 1;
  float *h_pi = (float*)malloc(size);
  *h_pi = 0;
  float *d_pi = NULL;
  err = hipMalloc((void **)&d_pi, size);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device d_pi (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }


  err = hipMemcpy(d_pi, h_pi, size, hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector pi from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  printf("Operaciones por Hilo %d\n",operacionPorHilo);
  calcularPi<<<blocksPerGrid, threadsPerBlock>>>(d_pi, operacionPorHilo);
  err = hipGetLastError();

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to launch calcularPi kernel (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_pi, d_pi, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        
        fprintf(stderr, "Failed to copy h_pi from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("valor de pi %f\n", (*h_pi)*4);
    return 0;

}