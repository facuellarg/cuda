#include "hip/hip_runtime.h"
#include "omp.h"
#include "stdio.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void
calcularPi(float *pi, int numElements, int operaciones)
{
	for(int j = 0; j < operaciones; j = j+4){
		int i = ((blockDim.x * blockIdx.x + threadIdx.x)*operaciones) + j + 1;
	      *pi+= 1.0/i;
        i +=2;
        *pi -= 1.0/i;	
	}
    
}
int main(void)
{
  // declarar  la cantidad de hilos segun la gpu
  hipError_t err = hipSuccess;
  int dev = 0;
  size_t size = sizeof(float);
	hipSetDevice(dev);
  hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	int threadsPerBlock = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
	threadsPerBlock = threadsPerBlock*2;
  int blocksPerGrid =   deviceProp.multiProcessorCount;
  int numIt = 4e8;
  int hilosTotales = blocksPerGrid*threadsPerBlock;
  int operacionPorHilo;
  operacionPorHilo = (numIt > hilosTotales ) ? (( numIt / hilosTotales ) + 1 ) : 1;
  float *h_pi = (float*)malloc(size);
  *h_pi = 0;
  float *d_pi = NULL;
  err = hipMalloc((void **)&d_pi, size);
  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device d_pi (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }


  err = hipMemcpy(d_pi, h_pi, size, hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector pi from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  printf("Operaciones por Hilo %d\n",operacionPorHilo);
  calcularPi<<<blocksPerGrid, threadsPerBlock>>>(d_pi, numIt,operacionPorHilo);
  err = hipGetLastError();

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to launch calcularPi kernel (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_pi, d_pi, sizeof(float), hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        
        fprintf(stderr, "Failed to copy h_pi from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    printf("valor de pi %f\n", (*h_pi)*4);
    return 0;

}